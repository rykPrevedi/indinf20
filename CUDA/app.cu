#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void funzione() {
	
	printf("\t\t\t[DEVICE] Hello, World!\n");
	
	int thrId = threadIdx.x;	// thread ID (monodimensional block)
	int blkId = blockIdx.x;		// block ID 
	
	int thrNum = blockDim.x;	// threads quantity 
	int blkNum = gridDim.x;		// blocks quantity
	printf("\t\t\t[DEVICE] Hello, World! I am thread #%d out of %d, and I belong to block #%d out of %d\n",
	thrId, 
	thrNum,
	blkId,
	blkNum);
	
	return;	
}

int main() {

	printf("[HOST] Hello, World!\n");
	
	funzione<<<3,5>>>();  // 3 block with 5 threads on GPU engine
	
	printf("[HOST] Some more work on host\n");
	
	hipDeviceSynchronize();
	
	printf("[HOST] Device ended its wotk!\n");
	
	return 0;
}