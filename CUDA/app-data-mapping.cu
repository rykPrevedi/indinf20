#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int arr[15];

__global__ void funzione() {
	
	printf("\t\t\t[DEVICE] Hello, World!\n");
	
	int thrId = threadIdx.x;	// thread ID (monodimensional block)
	int blkId = blockIdx.x;		// block ID 
	
	int thrNum = blockDim.x;	// threads quantity 
	int blkNum = gridDim.x;		// blocks quantity
	
	// data-mapping
	int i = (blkId * blockDim) + thrId;
	arr[i] = i * 2; 
	
	printf("\t\t\t[DEVICE] Hello, World! I am thread #%d out of %d, and I belong to block #%d out of %d\n",
	thrId, 
	thrNum,
	blkId,
	blkNum);
	

	
	return;	
}

int main() {

	printf("[HOST] Hello, World!\n");
	
	// kernel -> grid
	funzione<<<3,5>>>();  // 3 block with 5 threads
	
	printf("[HOST] Some more work on host\n");
	
	hipDeviceSynchronize();
	
	printf("[HOST] Device ended its wotk!\n");
	
	return 0;
}